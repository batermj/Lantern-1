#include "hip/hip_runtime.h"
#include <assert.h>
#include <err.h>
#include <errno.h>
#include <fcntl.h>
#include <functional>
#include <math.h>
#include <memory>
#include <random>
#include <stdint.h>
#include <stdio.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>
#include <cblas.h>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

long fsize(int fd) {
  struct stat stat;
  int res = fstat(fd, &stat);
  return stat.st_size;
}

int printll(char *s) {
  while (*s != '\n' && *s != ',' && *s != '\t') {
    putchar(*s++);
  }
  return 0;
}

long hash(char *str0, int len) {
  unsigned char *str = (unsigned char *)str0;
  unsigned long hash = 5381;
  int c;

  while ((c = *str++) && len--)
    hash = ((hash << 5) + hash) + c; /* hash * 33 + c */

  return hash;
}

long HEAP_SIZE_CPU = 1073741826; // 1048576; // 536870912; // 268435456; // 2097152; 1610612739; // 4294967304; //
void *mallocBase = calloc(HEAP_SIZE_CPU, 1);
void *mallocAddr = mallocBase;
void *waterMark = mallocBase;
void *myMalloc(size_t bytes) {
  void *res = mallocAddr;
  mallocAddr = (void *)((char *)mallocAddr + bytes);
  if ((long)mallocAddr >= (long)mallocBase + HEAP_SIZE_CPU)
    fprintf(stderr, "CPU memory breached limit of HEAP_SIZE_CPU\n");
  return res;
}

long HEAP_SIZE = 8589934608; //  4294967304; // this is for GPU

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1) {
  long int diff = (t2->tv_usec + 1000000 * t2->tv_sec) - (t1->tv_usec + 1000000 * t1->tv_sec);
  result->tv_sec = diff / 1000000;
  result->tv_usec = diff % 1000000;
  return (diff < 0);
}


#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    fprintf(stderr, "CUDA error occurred: %s (%s:%d)\n", \
            hipGetErrorString(err), __FILE__, __LINE__); \
    exit(err); \
  } \
}

#define CUBLAS_CALL(f) { \
  hipblasStatus_t stat = (f); \
  if (stat != HIPBLAS_STATUS_SUCCESS) { \
    fprintf(stderr, "cuBLAS error occurred: %d (%s:%d)\n", \
            stat, __FILE__, __LINE__); \
    exit(stat); \
  } \
}

void *gpuMallocBase;
void *gpuMallocAddr;

// Alignment boundary size, in bytes.
constexpr int N = 4; // 16
void *myGpuMalloc(size_t bytes) {
  bytes = ((bytes + (1 << N) - 1) >> N) << N;
  void *res = gpuMallocAddr;
  gpuMallocAddr = (void *)((char *)gpuMallocAddr + bytes);
  if ((long)gpuMallocAddr >= (long)gpuMallocBase + HEAP_SIZE)
    fprintf(stderr, "GPU breached memory limit of HEAP_SIZE\n");
  return res;
}

template <typename T>
__global__ void arrayUpdate(T *data, int index, T value) {
  data[index] = value;
}

__global__ void arrayFill_greg(float* data, float value, int size) {
  int stride = gridDim.x * blockDim.x;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < size; i += stride) data[i] = value;
}

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = tid; i < size; i += stride) {
    out[i] = in[i] < min_val ? min_val : (in[i] > max_val ? max_val : in[i]);
  }
}

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = tid; i < size; i += stride) {
    if (inplace) {
      if (in_x[i] < min_val || in_x[i] > max_val) in_d[i] = 0;
    } else {
      if (in_x[i] >= min_val && in_x[i] <= max_val) in_d[i] += out_d[i];
    }
  }
}

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = tid * x_stride + target[tid];
  y[tid] = -1 * x[offset];
}

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = tid * x_stride + target[tid];
  xGrad[offset] += -1 * yGrad[tid];
}

 // only for 4D tensor in and 3D tensor out
__global__ void sum_grad(float* in, int inSize0, int inSize1, int inSize2, int inSize3, int nElement,
                         float* out, int outStride0, int outStride1, int outStride2, int dim) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = tid; i < nElement; i += stride) {
    int inOff2 = i / inSize3;
    int inDim3 = i - inOff2 * inSize3;
    int inOff1 = inOff2 / inSize2;
    int inDim2 = inOff2 - inOff1 * inSize2;
    int inDim0 = inOff1 / inSize1;
    int inDim1 = inOff1 - inDim0 * inSize1;
    int outOff = 0;
    if (dim == 0) outOff = inDim1 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
    if (dim == 1) outOff = inDim0 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
    if (dim == 2) outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim3 * outStride2;
    if (dim == 3) outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim2 * outStride2;
    in[i] += out[outOff];
  }
}

//following - https://github.com/torch/cutorch/blob/master/lib/THC/THCTensorMath.cuh#L49
static inline __device__ int compute(int outputSize0, int outputSize1, int outputSize2, int outputSize3,
                                     int outputStride0, int outputStride1, int outputStride2, int outputStride3,
                                     const int dimSize, const int concatDim, int linearIndex) {
  int offset = 0;
  int curDimSize = 3 == concatDim ? dimSize : outputSize3;
  int nextDimIndex = linearIndex / curDimSize;
  int curDimIndex = linearIndex - curDimSize * nextDimIndex;
  int curDimOffset = curDimIndex * outputStride3;
  offset += curDimOffset;
  linearIndex = nextDimIndex;
  curDimSize = 2 == concatDim ? dimSize : outputSize2;
  nextDimIndex = linearIndex / curDimSize;
  curDimIndex = linearIndex - curDimSize * nextDimIndex;
  curDimOffset = curDimIndex * outputStride2;
  offset += curDimOffset;
  linearIndex = nextDimIndex;
  curDimSize = 1 == concatDim ? dimSize : outputSize1;
  nextDimIndex = linearIndex / curDimSize;
  curDimIndex = linearIndex - curDimSize * nextDimIndex;
  curDimOffset = curDimIndex * outputStride1;
  offset += curDimOffset;
  linearIndex = nextDimIndex;
  return offset + linearIndex * outputStride0;
//  for (int i = 3; i >= 1; i--) {
//    int curDimSize = i == concatDim ? dimSize : outputSize[i];
//    int nextDimIndex = linearIndex / curDimSize;
//    int curDimIndex = linearIndex - curDimSize * nextDimIndex;
//    int curDimOffset = curDimIndex * outputStride[i];
//    offset += curDimOffset;
//    linearIndex = nextDimIndex;
//  }
//  return offset + linearIndex * outputStride[0];
}

// TODO: Only for Dim of rank 4, and only for 2 inputs, and only for concat at dim = 1
__global__ void concat2D_1D_greg(float* in1, int dimSize1, int nElement1,
                                 float* in2, int dimSize2, int nElement2,
                                 float* out, int concatDim,
                                 int outSize0, int outSize1, int outSize2, int outSize3,
                                 int outStride0, int outStride1, int outStride2, int outStride3) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int nElement = blockIdx.y == 0 ? nElement1 : nElement2;
  if (tid >= nElement) return;
  float* data = blockIdx.y == 0 ? in1 : in2;
  int offset = blockIdx.y == 0 ? 0 : dimSize1;
  int dimSize = blockIdx.y == 0 ? dimSize1 : dimSize2;
  int dataOffset = offset * outStride1;
  int stride = gridDim.x * blockDim.x;
  while (tid < nElement) {
    int elementOffset = compute(outSize0, outSize1, outSize2, outSize3,
                                outStride0, outStride1, outStride2, outStride3, dimSize, concatDim, tid);
    out[dataOffset + elementOffset] = data[tid];
    tid += stride;
  }
}

// TODO: Only for Dim of rank 4, and only for 2 inputs, and only for concat at dim = 1
__global__ void concat2D_1D_greg_grad(float* in1, int dimSize1, int nElement1,
                                      float* in2, int dimSize2, int nElement2,
                                      float* out, int concatDim,
                                      int outSize0, int outSize1, int outSize2, int outSize3,
                                      int outStride0, int outStride1, int outStride2, int outStride3) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int nElement = blockIdx.y == 0 ? nElement1 : nElement2;
  if (tid >= nElement) return;
  float* data = blockIdx.y == 0 ? in1 : in2;
  int offset = blockIdx.y == 0 ? 0 : dimSize1;
  int dimSize = blockIdx.y == 0 ? dimSize1 : dimSize2;
  int dataOffset = offset * outStride1;
  int stride = gridDim.x * blockDim.x;
  while (tid < nElement) {
    int elementOffset = compute(outSize0, outSize1, outSize2, outSize3,
                                outStride0, outStride1, outStride2, outStride3, dimSize, concatDim, tid);
    data[tid] += out[dataOffset + elementOffset];
    tid += stride;
  }
}

__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < outScalarCount; tid += stride) {
    int linearIndex = tid;
    int outIndex0 = linearIndex / outStride0;
    linearIndex = linearIndex - outIndex0 * outStride0;
    int outIndex1 = linearIndex / outStride1;
    int outIndex2 = linearIndex - outIndex1 * outStride1;
    int inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
    out[tid] = in[inIndex];
  }
}

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < inScalarCount; tid += stride) {
    int linearIndex = tid;
    int inIndex0 = linearIndex / inStride0;
    linearIndex = linearIndex - inIndex0 * inStride0;
    int inIndex1 = linearIndex / inStride1;
    if (inIndex0 + inIndex1 >= inDim0) return;
    out[tid + inIndex1 * inStride0] = in[tid];
  }
}

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride) {
    if (d[tid] > clip) d[tid] = clip;
    if (d[tid] < -clip) d[tid] = -clip;
    m[tid] += d[tid] * d[tid];
    x[tid] -= lr * d[tid] / sqrt(m[tid] + 0.00000001);
    d[tid] = 0;
  }
}

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride) {
    float temp = d[tid];
    if (temp > gradClip) temp = gradClip;
    if (temp < -gradClip) temp = -gradClip;
    m[tid] *= momentum;
    m[tid] += temp;
    if (nesterov) { temp += momentum * m[tid]; }
    else { temp = m[tid]; }
    x[tid] -= learning_rate * temp;
    d[tid] = 0;
  }
}

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = in1[tid] * in2[tid];
}

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] += in1[tid] * in2[tid];
}

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = in1[tid] + in2[tid];
}

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = in1[tid] - in2[tid];
}

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = in1[tid] / in2[tid];
}

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = exp(in[tid]);
}
__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = log(in[tid]);
}
__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = sqrt(in[tid]);
}

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) out[tid] = in[tid] * in[tid];
}

__global__ void elementwise_1D_1D_exp_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) in_d[tid] += out_d[tid] * out_x[tid];
}

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) in_d[tid] += out_d[tid] / in_x[tid];
}

__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) in_d[tid] += out_d[tid] / out_x[tid] / 2;
}

__global__ void elementwise_1D_1D_square_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < size; tid += stride)
    if (tid < size) in_d[tid] += out_d[tid] * 2 * in_x[tid];
}

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < scalarCount; tid += stride) {
    int linearIndex = tid;
    int xindex0 = linearIndex / xstrides0;
    linearIndex = linearIndex - xstrides0 * xindex0;
    int xindex1 = linearIndex / xstrides1;
    linearIndex = linearIndex - xstrides1 * xindex1;
    int xindex2 = linearIndex / xstrides2;
    int xindex3 = linearIndex - xstrides2 * xindex2;
    if (xindex3 >= mask[xindex0]) in[tid] = 0;
  }
}

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < in1ScalarCount; tid += stride) {
    out[tid] = in1[tid] * in2[tid % in2ScalarCount];
  }
}

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (; tid < in1ScalarCount; tid += stride) {
    int index = tid % in2ScalarCount;
    in1_d[tid] += out[tid] * in2_x[index];
    in2_d[tid] = in1_x[tid] * out[tid];  // this is the temp array, need to be reduced!
  }
}

// From: https://github.com/pytorch/pytorch/blob/master/aten/src/THC/THCIntegerDivider.cuh
// Result of div/mod operation stored together.
template <typename Value>
struct DivMod {
  Value div, mod;

  __host__ __device__ DivMod(Value div, Value mod) : div(div), mod(mod) { }
};

// Base case: we only have an implementation for uint32_t for now.  For
// everything else, we use plain division.
template <typename Value>
struct IntDivider {
  IntDivider() { }  // Dummy constructor for arrays.
  IntDivider(Value d) : divisor(d) { }

  __host__ __device__ inline Value div(Value n) const { return n / divisor; }
  __host__ __device__ inline Value mod(Value n) const { return n % divisor; }
  __host__ __device__ inline DivMod<Value> divmod(Value n) const {
    return DivMod<Value>(n / divisor, n % divisor);
  }

  Value divisor;
};

// Implement fast integer division.
template <>
struct IntDivider<unsigned int> {
  static_assert(sizeof(unsigned int) == 4, "Assumes 32-bit unsigned int.");

  IntDivider() { }  // Dummy constructor for arrays.

  IntDivider(unsigned int d) : divisor(d) {
    assert(divisor >= 1 && divisor <= INT32_MAX);

    // TODO: gcc/clang has __builtin_clz() but it's not portable.
    for (shift = 0; shift < 32; shift++) if ((1U << shift) >= divisor) break;

    uint64_t one = 1;
    uint64_t magic = ((one << 32) * ((one << shift) - divisor)) / divisor + 1;
    m1 = magic;
    assert(m1 > 0 && m1 == magic);  // m1 must fit in 32 bits.
  }

  __host__ __device__ inline unsigned int div(unsigned int n) const {
#ifdef __CUDA_ARCH__
    // 't' is the higher 32-bits of unsigned 32-bit multiplication of 'n' and
    // 'm1'.
    unsigned int t = __umulhi(n, m1);
    return (t + n) >> shift;
#else
    // Using uint64_t so that the addition does not overflow.
    uint64_t t = ((uint64_t) n * m1) >> 32;
    return (t + n) >> shift;
#endif
  }

  __host__ __device__ inline unsigned int mod(unsigned int n) const {
    return n - div(n) * divisor;
  }

  __host__ __device__ inline DivMod<unsigned int> divmod(unsigned int n) const {
    unsigned int q = div(n);
    return DivMod<unsigned int>(q, n - q * divisor);
  }

  unsigned int divisor;  // d above.
  unsigned int m1;  // Magic number: m' above.
  unsigned int shift;  // Shift amounts.
};

// From: https://github.com/pytorch/pytorch/blob/master/aten/src/ATen/cuda/detail/OffsetCalculator.cuh
/// OffsetCalculator calculates the offset in bytes of a linear index for NARGS
/// operands that share the same shape, but may have different strides.

template <int NARGS>
struct OffsetCalculator {
  static constexpr int MAX_DIMS = 25;

  // The offset for each argument (in bytes). Wrapper around fixed-size array.
  struct offsets_t {
    __host__ __device__ uint32_t& operator[](int idx) {
      return values[idx];
    }
    uint32_t values[NARGS];
  };


  // OffsetCalculator(int dims, const int64_t* sizes, const int64_t* const* strides) : dims(dims) {
  OffsetCalculator(int dims, const int32_t* sizes, const int32_t* const* strides) : dims(dims) {
    for (int i = 0; i < MAX_DIMS; ++i) {
      if (i < dims) {
        sizes_[i] = IntDivider<uint32_t>(sizes[i]);
      } else {
        sizes_[i] = IntDivider<uint32_t>(1);
      }
      for (int arg = 0; arg < NARGS; arg++) {
        strides_[i][arg] = i < dims ? strides[arg][i] : 0;
      }
    }
  }

  __host__ __device__ offsets_t get(uint32_t linear_idx) const {
    offsets_t offsets;
#pragma unroll
    for (int arg = 0; arg < NARGS; arg++) {
      offsets[arg] = 0;
    }

#pragma unroll
    for (int dim = 0; dim < MAX_DIMS; ++dim) {
      if (dim == dims) {
        break;
      }
      auto divmod = sizes_[dim].divmod(linear_idx);
      linear_idx = divmod.div;

#pragma unroll
      for (int arg = 0; arg < NARGS; arg++) {
        offsets[arg] += divmod.mod * strides_[dim][arg];
      }
    }
    return offsets;
  }

  void print() {
    for (auto i = 1; i < 128; i++) {
      auto offsets = get(i);
      printf("offsets[%d]: ", i);
      for (auto arg = 0; arg < NARGS; arg++) {
        printf("%d ", offsets[arg]);
      }
      printf("\n");
    }
  }

  int dims;
  IntDivider<uint32_t> sizes_[MAX_DIMS];
  uint32_t strides_[MAX_DIMS][NARGS];
};

// From: https://github.com/pytorch/pytorch/blob/master/aten/src/ATen/native/cuda/Loops.cuh
template<int nt, int vt, typename func_t>
__launch_bounds__(nt, 4)
__global__ void elementwise_kernel(int N, func_t f) {
  int tid = threadIdx.x;
  int nv = nt * vt;
  int idx = nv * blockIdx.x + tid;
#pragma unroll
  for (int i = 0; i < vt; i++) {
    if (idx < N) {
      f(idx);
      idx += nt;
    }
  }
}

template<int nt, int vt, typename func_t>
static void launch_kernel(int64_t N, const func_t& f) {
  if (N == 0) {
    return;
  }
  dim3 block(nt);
  dim3 grid((N + block.x * vt - 1) / (block.x * vt));
  elementwise_kernel<nt, vt, func_t><<<grid, block, 0>>>(N, f);
}

template<typename func_t>
void gpu_unary_kernel(float *res, float *x,
                      int32_t resRank, const int32_t resScalarCount,
                      const int32_t* resShape, const int32_t* const* strides,
                      const func_t& f) {
  OffsetCalculator<2> calc(resRank, resShape, strides);
  launch_kernel<128, 4>(resScalarCount, [=]__device__(int idx) {
    auto offsets = calc.get(idx);
    float* out = &res[offsets[0]];
    float* in = &x[offsets[1]];
    *out = f(*in);
  });
}

template<typename func_t>
void gpu_binary_kernel(float *res, float *x, float *y,
                       int32_t resRank, const int32_t resScalarCount,
                       const int32_t* resShape, const int32_t* const* strides,
                       const func_t& f) {
  OffsetCalculator<3> calc(resRank, resShape, strides);
  launch_kernel<128, 4>(resScalarCount, [=]__device__(int idx) {
    auto offsets = calc.get(idx);
    float* out = &res[offsets[0]];
    float* in1 = &x[offsets[1]];
    float* in2 = &y[offsets[2]];
    *out = f(*in1, *in2);
  });
}

#define CUDNN_CALL(f) { \
  hipdnnStatus_t stat = (f); \
  if (stat != HIPDNN_STATUS_SUCCESS) { \
    fprintf(stderr, "cuDNN error occurred: %d (%s:%d)\n", \
            stat, __FILE__, __LINE__); \
    exit(stat); \
  } \
}


void Snippet(char *);

std::random_device rd{};
std::mt19937 gen{rd()};
std::normal_distribution<> d{0, 0.01};

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: query <filename>\n");
    return 0;
  }
  Snippet(argv[1]);
  return 0;
}

/*****************************************
  Emitting C Generated Code                  
*******************************************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
void Snippet(char*  x0) {
// Backend setup.
hipblasHandle_t cublasHandle;
CUBLAS_CALL(hipblasCreate(&cublasHandle));
CUDA_CALL(hipMalloc(&gpuMallocBase, HEAP_SIZE));
CUDA_CALL(hipMemset(gpuMallocBase, 0, HEAP_SIZE));
gpuMallocAddr = gpuMallocBase;
      
hipdnnHandle_t cudnnHandle;
CUDNN_CALL(hipdnnCreate(&cudnnHandle));
srand(42);
struct timeval begin_0, end_0, diff_0;
gettimeofday(&begin_0, NULL);
int32_t x7 = open("../../cifar10_data/cifar-10-batches-bin/data_batch_1.bin",0);
int64_t x8 = fsize(x7);
int64_t x10 = x8 / 3073LL;
int32_t x11 = (int32_t)x10;
int32_t x12 = x11 * 3072;
float* x13 = (float*)myMalloc(x12 * sizeof(float));;
int* x14 = (int32_t*)myMalloc(x11 * sizeof(int32_t));;
char* x9 = (char*)mmap(0, x8, PROT_READ | PROT_WRITE, MAP_FILE | MAP_PRIVATE, x7, 0);
for(int x16=0; x16 < x11; x16++) {
int32_t x17 = x16 * 3073;
char x18 = x9[x17];
int32_t x19 = (int32_t)(unsigned char)x18;
x14[x16] = x19;
int32_t x25 = x17 + 1;
int32_t x23 = x16 * 3072;
for(int x22=0; x22 < 3072; x22++) {
int32_t x26 = x25 + x22;
char x27 = x9[x26];
int32_t x24 = x23 + x22;
float x28 = (float)(unsigned char)x27;
float x29 = x28 / 255.0f;
x13[x24] = x29;

}

}
gettimeofday(&end_0, NULL);
timeval_subtract(&diff_0, &end_0, &begin_0);;
int64_t x37 = ((diff_0.tv_sec * 1000000L) + (diff_0.tv_usec));
float x38 = (float)x37;
float x39 = x38 / 1000000.0f;
printf("Data reading in %lf sec\n",x39);
// Tensor 'toGPU' invocation.
float* x98 = (float*)myGpuMalloc(32768 * sizeof(float));
int32_t x41 = open("/u/data/u99/wang603/TiarkMlEnv/Lantern/src/out/PLDI19evaluation/squeezenet/squeezenetCifar10.onnx.bin",0);
int64_t x42 = fsize(x41);
float* x43 = (float*)mmap(0, x42, PROT_READ | PROT_WRITE, MAP_FILE | MAP_PRIVATE, x41, 0);
float* x45 = x43+526720;
CUDA_CALL(hipMemcpy(x98, x45, 32768 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x101 = (float*)myGpuMalloc(48 * sizeof(float));
float* x46 = x43+245136;
CUDA_CALL(hipMemcpy(x101, x46, 48 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x104 = (float*)myGpuMalloc(64 * sizeof(float));
float* x47 = x43+17696;
CUDA_CALL(hipMemcpy(x104, x47, 64 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x107 = (float*)myGpuMalloc(81920 * sizeof(float));
float* x48 = x43+723904;
CUDA_CALL(hipMemcpy(x107, x48, 81920 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x110 = (float*)myGpuMalloc(64 * sizeof(float));
float* x49 = x43+14544;
CUDA_CALL(hipMemcpy(x110, x49, 64 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x113 = (float*)myGpuMalloc(36864 * sizeof(float));
float* x50 = x43+35392;
CUDA_CALL(hipMemcpy(x113, x50, 36864 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x116 = (float*)myGpuMalloc(4096 * sizeof(float));
float* x51 = x43+80608;
CUDA_CALL(hipMemcpy(x116, x51, 4096 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x119 = (float*)myGpuMalloc(16 * sizeof(float));
float* x52 = x43+4224;
CUDA_CALL(hipMemcpy(x119, x52, 16 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x122 = (float*)myGpuMalloc(64 * sizeof(float));
float* x53 = x43+362304;
CUDA_CALL(hipMemcpy(x122, x53, 64 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x125 = (float*)myGpuMalloc(4096 * sizeof(float));
float* x54 = x43+27040;
CUDA_CALL(hipMemcpy(x125, x54, 4096 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x128 = (float*)myGpuMalloc(1024 * sizeof(float));
float* x55 = x43+16672;
CUDA_CALL(hipMemcpy(x128, x55, 1024 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x131 = (float*)myGpuMalloc(2048 * sizeof(float));
float* x56 = x43+14608;
CUDA_CALL(hipMemcpy(x131, x56, 2048 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x134 = (float*)myGpuMalloc(256 * sizeof(float));
float* x57 = x43+526464;
CUDA_CALL(hipMemcpy(x134, x57, 256 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x137 = (float*)myGpuMalloc(18432 * sizeof(float));
float* x58 = x43+226704;
CUDA_CALL(hipMemcpy(x137, x58, 18432 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x140 = (float*)myGpuMalloc(32 * sizeof(float));
float* x59 = x43+80576;
CUDA_CALL(hipMemcpy(x140, x59, 32 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x143 = (float*)myGpuMalloc(128 * sizeof(float));
float* x60 = x43+121696;
CUDA_CALL(hipMemcpy(x143, x60, 128 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x146 = (float*)myGpuMalloc(256 * sizeof(float));
float* x61 = x43+723648;
CUDA_CALL(hipMemcpy(x146, x61, 256 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x149 = (float*)myGpuMalloc(82944 * sizeof(float));
float* x62 = x43+254592;
CUDA_CALL(hipMemcpy(x149, x62, 82944 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x152 = (float*)myGpuMalloc(9216 * sizeof(float));
float* x63 = x43+17760;
CUDA_CALL(hipMemcpy(x152, x63, 9216 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x155 = (float*)myGpuMalloc(64 * sizeof(float));
float* x64 = x43+559488;
CUDA_CALL(hipMemcpy(x155, x64, 64 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x158 = (float*)myGpuMalloc(128 * sizeof(float));
float* x65 = x43+84704;
CUDA_CALL(hipMemcpy(x158, x65, 128 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x161 = (float*)myGpuMalloc(9216 * sizeof(float));
float* x66 = x43+245184;
CUDA_CALL(hipMemcpy(x161, x66, 9216 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x164 = (float*)myGpuMalloc(32 * sizeof(float));
float* x67 = x43+31136;
CUDA_CALL(hipMemcpy(x164, x67, 32 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x167 = (float*)myGpuMalloc(1024 * sizeof(float));
float* x68 = x43+4240;
CUDA_CALL(hipMemcpy(x167, x68, 1024 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x170 = (float*)myGpuMalloc(16 * sizeof(float));
float* x69 = x43+16656;
CUDA_CALL(hipMemcpy(x170, x69, 16 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x173 = (float*)myGpuMalloc(256 * sizeof(float));
float* x70 = x43+575936;
CUDA_CALL(hipMemcpy(x173, x70, 256 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x176 = (float*)myGpuMalloc(8192 * sizeof(float));
float* x71 = x43+72384;
CUDA_CALL(hipMemcpy(x176, x71, 8192 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x179 = (float*)myGpuMalloc(147456 * sizeof(float));
float* x72 = x43+379008;
CUDA_CALL(hipMemcpy(x179, x72, 147456 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x182 = (float*)myGpuMalloc(192 * sizeof(float));
float* x73 = x43+226512;
CUDA_CALL(hipMemcpy(x182, x73, 192 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x185 = (float*)myGpuMalloc(147456 * sizeof(float));
float* x74 = x43+576192;
CUDA_CALL(hipMemcpy(x185, x74, 147456 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x188 = (float*)myGpuMalloc(64 * sizeof(float));
float* x75 = x43+5264;
CUDA_CALL(hipMemcpy(x188, x75, 64 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x191 = (float*)myGpuMalloc(192 * sizeof(float));
float* x76 = x43+254400;
CUDA_CALL(hipMemcpy(x191, x76, 192 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x194 = (float*)myGpuMalloc(2592 * sizeof(float));
float* x77 = x43+0;
CUDA_CALL(hipMemcpy(x194, x77, 2592 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x197 = (float*)myGpuMalloc(24576 * sizeof(float));
float* x78 = x43+337728;
CUDA_CALL(hipMemcpy(x197, x78, 24576 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x200 = (float*)myGpuMalloc(4096 * sizeof(float));
float* x79 = x43+31168;
CUDA_CALL(hipMemcpy(x200, x79, 4096 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x203 = (float*)myGpuMalloc(36864 * sizeof(float));
float* x80 = x43+84832;
CUDA_CALL(hipMemcpy(x203, x80, 36864 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x206 = (float*)myGpuMalloc(64 * sizeof(float));
float* x81 = x43+26976;
CUDA_CALL(hipMemcpy(x206, x81, 64 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x209 = (float*)myGpuMalloc(16384 * sizeof(float));
float* x82 = x43+559552;
CUDA_CALL(hipMemcpy(x209, x82, 16384 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x212 = (float*)myGpuMalloc(82944 * sizeof(float));
float* x83 = x43+143568;
CUDA_CALL(hipMemcpy(x212, x83, 82944 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x215 = (float*)myGpuMalloc(256 * sizeof(float));
float* x84 = x43+378752;
CUDA_CALL(hipMemcpy(x215, x84, 256 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x218 = (float*)myGpuMalloc(128 * sizeof(float));
float* x85 = x43+72256;
CUDA_CALL(hipMemcpy(x218, x85, 128 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x221 = (float*)myGpuMalloc(12288 * sizeof(float));
float* x86 = x43+121824;
CUDA_CALL(hipMemcpy(x221, x86, 12288 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x224 = (float*)myGpuMalloc(96 * sizeof(float));
float* x87 = x43+2592;
CUDA_CALL(hipMemcpy(x224, x87, 96 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x227 = (float*)myGpuMalloc(192 * sizeof(float));
float* x88 = x43+337536;
CUDA_CALL(hipMemcpy(x227, x88, 192 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x230 = (float*)myGpuMalloc(128 * sizeof(float));
float* x89 = x43+35264;
CUDA_CALL(hipMemcpy(x230, x89, 128 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x233 = (float*)myGpuMalloc(192 * sizeof(float));
float* x90 = x43+143376;
CUDA_CALL(hipMemcpy(x233, x90, 192 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x236 = (float*)myGpuMalloc(9216 * sizeof(float));
float* x91 = x43+5328;
CUDA_CALL(hipMemcpy(x236, x91, 9216 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x239 = (float*)myGpuMalloc(9216 * sizeof(float));
float* x92 = x43+134160;
CUDA_CALL(hipMemcpy(x239, x92, 9216 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x242 = (float*)myGpuMalloc(16384 * sizeof(float));
float* x93 = x43+362368;
CUDA_CALL(hipMemcpy(x242, x93, 16384 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x245 = (float*)myGpuMalloc(1536 * sizeof(float));
float* x94 = x43+2688;
CUDA_CALL(hipMemcpy(x245, x94, 1536 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x248 = (float*)myGpuMalloc(10 * sizeof(float));
float* x95 = x43+805824;
CUDA_CALL(hipMemcpy(x248, x95, 10 * sizeof(float), hipMemcpyHostToDevice));
// Tensor 'toGPU' invocation.
float* x251 = (float*)myGpuMalloc(48 * sizeof(float));
float* x96 = x43+134112;
CUDA_CALL(hipMemcpy(x251, x96, 48 * sizeof(float), hipMemcpyHostToDevice));
int64_t x253 = (long)mallocAddr;
int64_t x254 = (long)gpuMallocAddr;
// inferencing loop starts here
int32_t x262 = x11 / 64;
bool x272 = 34 >= 3;
bool x273;
if (x272) {
x273 = x272;
} else {
x273 = false;
}
int32_t x278 = 31 / 1;
int32_t x279 = x278 + 1;
int32_t x283 = 6144 * x279;
int32_t x284 = x283 * x279;
int32_t x280 = x279 * x279;
int32_t x281 = 96 * x280;
int32_t x282 = 64 * x281;
int32_t x306 = x279 - 2;
int32_t x307 = x306 / 2;
int32_t x308 = x307 + 1;
int32_t x312 = 6144 * x308;
int32_t x313 = x312 * x308;
bool x316 = x308 >= 1;
bool x317;
if (x316) {
x317 = x316;
} else {
x317 = false;
}
int32_t x322 = x307 / 1;
int32_t x323 = x322 + 1;
int32_t x327 = 1024 * x323;
int32_t x328 = x327 * x323;
int32_t x324 = x323 * x323;
int32_t x325 = 16 * x324;
int32_t x326 = 64 * x325;
bool x346 = x323 >= 1;
bool x347;
if (x346) {
x347 = x346;
} else {
x347 = false;
}
int32_t x352 = x322 / 1;
int32_t x353 = x352 + 1;
int32_t x357 = 4096 * x353;
int32_t x358 = x357 * x353;
int32_t x354 = x353 * x353;
int32_t x355 = 64 * x354;
int32_t x356 = 64 * x355;
int32_t x376 = x323 + 2;
bool x377 = x376 >= 3;
bool x378;
if (x377) {
x378 = x377;
} else {
x378 = false;
}
int32_t x383 = x376 - 3;
int32_t x384 = x383 / 1;
int32_t x385 = x384 + 1;
int32_t x389 = 4096 * x385;
int32_t x390 = x389 * x385;
int32_t x386 = x385 * x385;
int32_t x387 = 64 * x386;
int32_t x388 = 64 * x387;
bool x408 = true || false;
bool x410;
if (x408) {
bool x409 = true || true;
x410 = x409;
} else {
x410 = false;
}
bool x413;
if (x410) {
bool x411 = x385 == x353;
bool x412 = x411 || false;
x413 = x412;
} else {
x413 = false;
}
bool x414;
if (x413) {
bool x411 = x385 == x353;
bool x412 = x411 || false;
x414 = x412;
} else {
x414 = false;
}
int32_t x423 = 8192 * x353;
int32_t x424 = x423 * x353;
int32_t x421 = 128 * x354;
bool x427 = x353 >= 1;
bool x428;
if (x427) {
x428 = x427;
} else {
x428 = false;
}
int32_t x433 = x352 / 1;
int32_t x434 = x433 + 1;
int32_t x438 = 1024 * x434;
int32_t x439 = x438 * x434;
int32_t x435 = x434 * x434;
int32_t x436 = 16 * x435;
int32_t x437 = 64 * x436;
bool x457 = x434 >= 1;
bool x458;
if (x457) {
x458 = x457;
} else {
x458 = false;
}
int32_t x463 = x433 / 1;
int32_t x464 = x463 + 1;
int32_t x468 = 4096 * x464;
int32_t x469 = x468 * x464;
int32_t x465 = x464 * x464;
int32_t x466 = 64 * x465;
int32_t x467 = 64 * x466;
int32_t x487 = x434 + 2;
bool x488 = x487 >= 3;
bool x489;
if (x488) {
x489 = x488;
} else {
x489 = false;
}
int32_t x494 = x487 - 3;
int32_t x495 = x494 / 1;
int32_t x496 = x495 + 1;
int32_t x500 = 4096 * x496;
int32_t x501 = x500 * x496;
int32_t x497 = x496 * x496;
int32_t x498 = 64 * x497;
int32_t x499 = 64 * x498;
bool x521;
if (x410) {
bool x519 = x496 == x464;
bool x520 = x519 || false;
x521 = x520;
} else {
x521 = false;
}
bool x522;
if (x521) {
bool x519 = x496 == x464;
bool x520 = x519 || false;
x522 = x520;
} else {
x522 = false;
}
int32_t x531 = 8192 * x464;
int32_t x532 = x531 * x464;
int32_t x529 = 128 * x465;
bool x535 = x464 >= 1;
bool x536;
if (x535) {
x536 = x535;
} else {
x536 = false;
}
int32_t x541 = x463 / 1;
int32_t x542 = x541 + 1;
int32_t x546 = 2048 * x542;
int32_t x547 = x546 * x542;
int32_t x543 = x542 * x542;
int32_t x544 = 32 * x543;
int32_t x545 = 64 * x544;
bool x565 = x542 >= 1;
bool x566;
if (x565) {
x566 = x565;
} else {
x566 = false;
}
int32_t x571 = x541 / 1;
int32_t x572 = x571 + 1;
int32_t x576 = 8192 * x572;
int32_t x577 = x576 * x572;
int32_t x573 = x572 * x572;
int32_t x574 = 128 * x573;
int32_t x575 = 64 * x574;
int32_t x595 = x542 + 2;
bool x596 = x595 >= 3;
bool x597;
if (x596) {
x597 = x596;
} else {
x597 = false;
}
int32_t x602 = x595 - 3;
int32_t x603 = x602 / 1;
int32_t x604 = x603 + 1;
int32_t x608 = 8192 * x604;
int32_t x609 = x608 * x604;
int32_t x605 = x604 * x604;
int32_t x606 = 128 * x605;
int32_t x607 = 64 * x606;
bool x629;
if (x410) {
bool x627 = x604 == x572;
bool x628 = x627 || false;
x629 = x628;
} else {
x629 = false;
}
bool x630;
if (x629) {
bool x627 = x604 == x572;
bool x628 = x627 || false;
x630 = x628;
} else {
x630 = false;
}
int32_t x639 = 16384 * x572;
int32_t x640 = x639 * x572;
int32_t x637 = 256 * x573;
int32_t x647 = x572 - 2;
int32_t x648 = x647 / 2;
int32_t x649 = x648 + 1;
int32_t x653 = 16384 * x649;
int32_t x654 = x653 * x649;
bool x657 = x649 >= 1;
bool x658;
if (x657) {
x658 = x657;
} else {
x658 = false;
}
int32_t x663 = x648 / 1;
int32_t x664 = x663 + 1;
int32_t x668 = 2048 * x664;
int32_t x669 = x668 * x664;
int32_t x665 = x664 * x664;
int32_t x666 = 32 * x665;
int32_t x667 = 64 * x666;
bool x687 = x664 >= 1;
bool x688;
if (x687) {
x688 = x687;
} else {
x688 = false;
}
int32_t x693 = x663 / 1;
int32_t x694 = x693 + 1;
int32_t x698 = 8192 * x694;
int32_t x699 = x698 * x694;
int32_t x695 = x694 * x694;
int32_t x696 = 128 * x695;
int32_t x697 = 64 * x696;
int32_t x717 = x664 + 2;
bool x718 = x717 >= 3;
bool x719;
if (x718) {
x719 = x718;
} else {
x719 = false;
}
int32_t x724 = x717 - 3;
int32_t x725 = x724 / 1;
int32_t x726 = x725 + 1;
int32_t x730 = 8192 * x726;
int32_t x731 = x730 * x726;
int32_t x727 = x726 * x726;
int32_t x728 = 128 * x727;
int32_t x729 = 64 * x728;
bool x751;
if (x410) {
bool x749 = x726 == x694;
bool x750 = x749 || false;
x751 = x750;
} else {
x751 = false;
}
bool x752;
if (x751) {
bool x749 = x726 == x694;
bool x750 = x749 || false;
x752 = x750;
} else {
x752 = false;
}
int32_t x761 = 16384 * x694;
int32_t x762 = x761 * x694;
int32_t x759 = 256 * x695;
bool x765 = x694 >= 1;
bool x766;
if (x765) {
x766 = x765;
} else {
x766 = false;
}
int32_t x771 = x693 / 1;
int32_t x772 = x771 + 1;
int32_t x776 = 3072 * x772;
int32_t x777 = x776 * x772;
int32_t x773 = x772 * x772;
int32_t x774 = 48 * x773;
int32_t x775 = 64 * x774;
bool x795 = x772 >= 1;
bool x796;
if (x795) {
x796 = x795;
} else {
x796 = false;
}
int32_t x801 = x771 / 1;
int32_t x802 = x801 + 1;
int32_t x806 = 12288 * x802;
int32_t x807 = x806 * x802;
int32_t x803 = x802 * x802;
int32_t x804 = 192 * x803;
int32_t x805 = 64 * x804;
int32_t x825 = x772 + 2;
bool x826 = x825 >= 3;
bool x827;
if (x826) {
x827 = x826;
} else {
x827 = false;
}
int32_t x832 = x825 - 3;
int32_t x833 = x832 / 1;
int32_t x834 = x833 + 1;
int32_t x838 = 12288 * x834;
int32_t x839 = x838 * x834;
int32_t x835 = x834 * x834;
int32_t x836 = 192 * x835;
int32_t x837 = 64 * x836;
bool x859;
if (x410) {
bool x857 = x834 == x802;
bool x858 = x857 || false;
x859 = x858;
} else {
x859 = false;
}
bool x860;
if (x859) {
bool x857 = x834 == x802;
bool x858 = x857 || false;
x860 = x858;
} else {
x860 = false;
}
int32_t x869 = 24576 * x802;
int32_t x870 = x869 * x802;
int32_t x867 = 384 * x803;
bool x873 = x802 >= 1;
bool x874;
if (x873) {
x874 = x873;
} else {
x874 = false;
}
int32_t x879 = x801 / 1;
int32_t x880 = x879 + 1;
int32_t x884 = 3072 * x880;
int32_t x885 = x884 * x880;
int32_t x881 = x880 * x880;
int32_t x882 = 48 * x881;
int32_t x883 = 64 * x882;
bool x903 = x880 >= 1;
bool x904;
if (x903) {
x904 = x903;
} else {
x904 = false;
}
int32_t x909 = x879 / 1;
int32_t x910 = x909 + 1;
int32_t x914 = 12288 * x910;
int32_t x915 = x914 * x910;
int32_t x911 = x910 * x910;
int32_t x912 = 192 * x911;
int32_t x913 = 64 * x912;
int32_t x933 = x880 + 2;
bool x934 = x933 >= 3;
bool x935;
if (x934) {
x935 = x934;
} else {
x935 = false;
}
int32_t x940 = x933 - 3;
int32_t x941 = x940 / 1;
int32_t x942 = x941 + 1;
int32_t x946 = 12288 * x942;
int32_t x947 = x946 * x942;
int32_t x943 = x942 * x942;
int32_t x944 = 192 * x943;
int32_t x945 = 64 * x944;
bool x967;
if (x410) {
bool x965 = x942 == x910;
bool x966 = x965 || false;
x967 = x966;
} else {
x967 = false;
}
bool x968;
if (x967) {
bool x965 = x942 == x910;
bool x966 = x965 || false;
x968 = x966;
} else {
x968 = false;
}
int32_t x977 = 24576 * x910;
int32_t x978 = x977 * x910;
int32_t x975 = 384 * x911;
bool x981 = x910 >= 1;
bool x982;
if (x981) {
x982 = x981;
} else {
x982 = false;
}
int32_t x987 = x909 / 1;
int32_t x988 = x987 + 1;
int32_t x992 = 4096 * x988;
int32_t x993 = x992 * x988;
int32_t x989 = x988 * x988;
int32_t x990 = 64 * x989;
int32_t x991 = 64 * x990;
bool x1011 = x988 >= 1;
bool x1012;
if (x1011) {
x1012 = x1011;
} else {
x1012 = false;
}
int32_t x1017 = x987 / 1;
int32_t x1018 = x1017 + 1;
int32_t x1022 = 16384 * x1018;
int32_t x1023 = x1022 * x1018;
int32_t x1019 = x1018 * x1018;
int32_t x1020 = 256 * x1019;
int32_t x1021 = 64 * x1020;
int32_t x1041 = x988 + 2;
bool x1042 = x1041 >= 3;
bool x1043;
if (x1042) {
x1043 = x1042;
} else {
x1043 = false;
}
int32_t x1048 = x1041 - 3;
int32_t x1049 = x1048 / 1;
int32_t x1050 = x1049 + 1;
int32_t x1054 = 16384 * x1050;
int32_t x1055 = x1054 * x1050;
int32_t x1051 = x1050 * x1050;
int32_t x1052 = 256 * x1051;
int32_t x1053 = 64 * x1052;
bool x1075;
if (x410) {
bool x1073 = x1050 == x1018;
bool x1074 = x1073 || false;
x1075 = x1074;
} else {
x1075 = false;
}
bool x1076;
if (x1075) {
bool x1073 = x1050 == x1018;
bool x1074 = x1073 || false;
x1076 = x1074;
} else {
x1076 = false;
}
int32_t x1085 = 32768 * x1018;
int32_t x1086 = x1085 * x1018;
int32_t x1083 = 512 * x1019;
int32_t x1093 = x1018 - 2;
int32_t x1094 = x1093 / 2;
int32_t x1095 = x1094 + 1;
int32_t x1099 = 32768 * x1095;
int32_t x1100 = x1099 * x1095;
bool x1103 = x1095 >= 1;
bool x1104;
if (x1103) {
x1104 = x1103;
} else {
x1104 = false;
}
int32_t x1109 = x1094 / 1;
int32_t x1110 = x1109 + 1;
int32_t x1114 = 4096 * x1110;
int32_t x1115 = x1114 * x1110;
int32_t x1111 = x1110 * x1110;
int32_t x1112 = 64 * x1111;
int32_t x1113 = 64 * x1112;
bool x1133 = x1110 >= 1;
bool x1134;
if (x1133) {
x1134 = x1133;
} else {
x1134 = false;
}
int32_t x1139 = x1109 / 1;
int32_t x1140 = x1139 + 1;
int32_t x1144 = 16384 * x1140;
int32_t x1145 = x1144 * x1140;
int32_t x1141 = x1140 * x1140;
int32_t x1142 = 256 * x1141;
int32_t x1143 = 64 * x1142;
int32_t x1163 = x1110 + 2;
bool x1164 = x1163 >= 3;
bool x1165;
if (x1164) {
x1165 = x1164;
} else {
x1165 = false;
}
int32_t x1170 = x1163 - 3;
int32_t x1171 = x1170 / 1;
int32_t x1172 = x1171 + 1;
int32_t x1176 = 16384 * x1172;
int32_t x1177 = x1176 * x1172;
int32_t x1173 = x1172 * x1172;
int32_t x1174 = 256 * x1173;
int32_t x1175 = 64 * x1174;
bool x1197;
if (x410) {
bool x1195 = x1172 == x1140;
bool x1196 = x1195 || false;
x1197 = x1196;
} else {
x1197 = false;
}
bool x1198;
if (x1197) {
bool x1195 = x1172 == x1140;
bool x1196 = x1195 || false;
x1198 = x1196;
} else {
x1198 = false;
}
int32_t x1207 = 32768 * x1140;
int32_t x1208 = x1207 * x1140;
int32_t x1205 = 512 * x1141;
bool x1211 = x1140 >= 4;
bool x1212;
if (x1211) {
x1212 = x1211;
} else {
x1212 = false;
}
int32_t x1217 = x1140 - 4;
int32_t x1218 = x1217 / 1;
int32_t x1219 = x1218 + 1;
int32_t x1223 = 640 * x1219;
int32_t x1224 = x1223 * x1219;
int64_t x1250 = (int64_t)x11;
for(int x257=0; x257 < 4; x257++) {
struct timeval begin_1, end_1, diff_1;
int32_t x259 = x257 + 1;
printf("Start inferencing epoch %d\n",x259);
gettimeofday(&begin_1, NULL);
for(int x264=0; x264 < x262; x264++) {
int32_t x265 = x264 * 64;
int32_t x266 = x265 * 3072;
float* x267 = x13+x266;
int* x268 = x14+x265;
// Tensor 'toGPU' invocation.
float* x270 = (float*)myGpuMalloc(196608 * sizeof(float));
CUDA_CALL(hipMemcpy(x270, x267, 196608 * sizeof(float), hipMemcpyHostToDevice));
if (x273) {
} else {
assert(false && "ERROR not specified");
}
float* x285 = (float*)myGpuMalloc(x284 * sizeof(float));
float* x286 = (float*)myMalloc(1 * sizeof(float));;
x286[0] = 0.0f;
float* x288 = (float*)myMalloc(1 * sizeof(float));;
x288[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 3, 32, 32));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    96, 3, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 96, x279, x279));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x288, in_desc, x270, filt_desc, x194,
    conv_desc, algo, ws_data, ws_size,
    x286, out_desc, x285));
};
float* x291 = (float*)myMalloc(1 * sizeof(float));;
x291[0] = 1.0f;
float* x293 = (float*)myMalloc(1 * sizeof(float));;
x293[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 96, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 96, x279, x279));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x291, bias_desc, x224, x293, out_desc, x285));
};
float* x296 = (float*)myMalloc(1 * sizeof(float));;
x296[0] = 0.0f;
float* x298 = (float*)myMalloc(1 * sizeof(float));;
x298[0] = 1.0f;
float* x300 = (float*)myGpuMalloc(x282 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 96, x279, x279));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x298, x_desc, x285, x296, x_desc, x300));
};
float* x302 = (float*)myMalloc(1 * sizeof(float));;
x302[0] = 0.0f;
float* x304 = (float*)myMalloc(1 * sizeof(float));;
x304[0] = 1.0f;
float* x314 = (float*)myGpuMalloc(x313 * sizeof(float));

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 96, x279, x279) );

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 96, x308, x308));

hipdnnPoolingDescriptor_t poolingDesc;
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&poolingDesc));
CUDNN_CALL(hipdnnSetPooling2dDescriptor(
    poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
    2, 2, 0,
    0, 2, 2
));
CUDNN_CALL(hipdnnPoolingForward(
    cudnnHandle, 
    poolingDesc, 
    x304, in_desc, x300, x302, out_desc, x314));
};
if (x317) {
} else {
assert(false && "ERROR not specified");
}
float* x329 = (float*)myGpuMalloc(x328 * sizeof(float));
float* x330 = (float*)myMalloc(1 * sizeof(float));;
x330[0] = 0.0f;
float* x332 = (float*)myMalloc(1 * sizeof(float));;
x332[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 96, x308, x308));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    16, 96, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x323, x323));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x332, in_desc, x314, filt_desc, x245,
    conv_desc, algo, ws_data, ws_size,
    x330, out_desc, x329));
};
float* x335 = (float*)myMalloc(1 * sizeof(float));;
x335[0] = 1.0f;
float* x337 = (float*)myMalloc(1 * sizeof(float));;
x337[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 16, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x323, x323));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x335, bias_desc, x119, x337, out_desc, x329));
};
float* x340 = (float*)myMalloc(1 * sizeof(float));;
x340[0] = 0.0f;
float* x342 = (float*)myMalloc(1 * sizeof(float));;
x342[0] = 1.0f;
float* x344 = (float*)myGpuMalloc(x326 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x323, x323));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x342, x_desc, x329, x340, x_desc, x344));
};
if (x347) {
} else {
assert(false && "ERROR not specified");
}
float* x359 = (float*)myGpuMalloc(x358 * sizeof(float));
float* x360 = (float*)myMalloc(1 * sizeof(float));;
x360[0] = 0.0f;
float* x362 = (float*)myMalloc(1 * sizeof(float));;
x362[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x323, x323));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    64, 16, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x353, x353));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x362, in_desc, x344, filt_desc, x167,
    conv_desc, algo, ws_data, ws_size,
    x360, out_desc, x359));
};
float* x365 = (float*)myMalloc(1 * sizeof(float));;
x365[0] = 1.0f;
float* x367 = (float*)myMalloc(1 * sizeof(float));;
x367[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x353, x353));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x365, bias_desc, x188, x367, out_desc, x359));
};
float* x370 = (float*)myMalloc(1 * sizeof(float));;
x370[0] = 0.0f;
float* x372 = (float*)myMalloc(1 * sizeof(float));;
x372[0] = 1.0f;
float* x374 = (float*)myGpuMalloc(x356 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x353, x353));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x372, x_desc, x359, x370, x_desc, x374));
};
if (x378) {
} else {
assert(false && "ERROR not specified");
}
float* x391 = (float*)myGpuMalloc(x390 * sizeof(float));
float* x392 = (float*)myMalloc(1 * sizeof(float));;
x392[0] = 0.0f;
float* x394 = (float*)myMalloc(1 * sizeof(float));;
x394[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x323, x323));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    64, 16, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x385, x385));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x394, in_desc, x344, filt_desc, x236,
    conv_desc, algo, ws_data, ws_size,
    x392, out_desc, x391));
};
float* x397 = (float*)myMalloc(1 * sizeof(float));;
x397[0] = 1.0f;
float* x399 = (float*)myMalloc(1 * sizeof(float));;
x399[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x385, x385));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x397, bias_desc, x110, x399, out_desc, x391));
};
float* x402 = (float*)myMalloc(1 * sizeof(float));;
x402[0] = 0.0f;
float* x404 = (float*)myMalloc(1 * sizeof(float));;
x404[0] = 1.0f;
float* x406 = (float*)myGpuMalloc(x388 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x385, x385));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x404, x_desc, x391, x402, x_desc, x406));
};
if (x414) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x425 = (float*)myGpuMalloc(x424 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x374, 64, x356, x406, 64, x388, x425, 1, 64, 128, x353, x353, x421, x354, x353, 1);
};
if (x428) {
} else {
assert(false && "ERROR not specified");
}
float* x440 = (float*)myGpuMalloc(x439 * sizeof(float));
float* x441 = (float*)myMalloc(1 * sizeof(float));;
x441[0] = 0.0f;
float* x443 = (float*)myMalloc(1 * sizeof(float));;
x443[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x353, x353));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    16, 128, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x434, x434));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x443, in_desc, x425, filt_desc, x131,
    conv_desc, algo, ws_data, ws_size,
    x441, out_desc, x440));
};
float* x446 = (float*)myMalloc(1 * sizeof(float));;
x446[0] = 1.0f;
float* x448 = (float*)myMalloc(1 * sizeof(float));;
x448[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 16, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x434, x434));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x446, bias_desc, x170, x448, out_desc, x440));
};
float* x451 = (float*)myMalloc(1 * sizeof(float));;
x451[0] = 0.0f;
float* x453 = (float*)myMalloc(1 * sizeof(float));;
x453[0] = 1.0f;
float* x455 = (float*)myGpuMalloc(x437 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x434, x434));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x453, x_desc, x440, x451, x_desc, x455));
};
if (x458) {
} else {
assert(false && "ERROR not specified");
}
float* x470 = (float*)myGpuMalloc(x469 * sizeof(float));
float* x471 = (float*)myMalloc(1 * sizeof(float));;
x471[0] = 0.0f;
float* x473 = (float*)myMalloc(1 * sizeof(float));;
x473[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x434, x434));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    64, 16, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x464, x464));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x473, in_desc, x455, filt_desc, x128,
    conv_desc, algo, ws_data, ws_size,
    x471, out_desc, x470));
};
float* x476 = (float*)myMalloc(1 * sizeof(float));;
x476[0] = 1.0f;
float* x478 = (float*)myMalloc(1 * sizeof(float));;
x478[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x464, x464));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x476, bias_desc, x104, x478, out_desc, x470));
};
float* x481 = (float*)myMalloc(1 * sizeof(float));;
x481[0] = 0.0f;
float* x483 = (float*)myMalloc(1 * sizeof(float));;
x483[0] = 1.0f;
float* x485 = (float*)myGpuMalloc(x467 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x464, x464));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x483, x_desc, x470, x481, x_desc, x485));
};
if (x489) {
} else {
assert(false && "ERROR not specified");
}
float* x502 = (float*)myGpuMalloc(x501 * sizeof(float));
float* x503 = (float*)myMalloc(1 * sizeof(float));;
x503[0] = 0.0f;
float* x505 = (float*)myMalloc(1 * sizeof(float));;
x505[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 16, x434, x434));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    64, 16, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x496, x496));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x505, in_desc, x455, filt_desc, x152,
    conv_desc, algo, ws_data, ws_size,
    x503, out_desc, x502));
};
float* x508 = (float*)myMalloc(1 * sizeof(float));;
x508[0] = 1.0f;
float* x510 = (float*)myMalloc(1 * sizeof(float));;
x510[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x496, x496));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x508, bias_desc, x206, x510, out_desc, x502));
};
float* x513 = (float*)myMalloc(1 * sizeof(float));;
x513[0] = 0.0f;
float* x515 = (float*)myMalloc(1 * sizeof(float));;
x515[0] = 1.0f;
float* x517 = (float*)myGpuMalloc(x499 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x496, x496));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x515, x_desc, x502, x513, x_desc, x517));
};
if (x522) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x533 = (float*)myGpuMalloc(x532 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x485, 64, x467, x517, 64, x499, x533, 1, 64, 128, x464, x464, x529, x465, x464, 1);
};
if (x536) {
} else {
assert(false && "ERROR not specified");
}
float* x548 = (float*)myGpuMalloc(x547 * sizeof(float));
float* x549 = (float*)myMalloc(1 * sizeof(float));;
x549[0] = 0.0f;
float* x551 = (float*)myMalloc(1 * sizeof(float));;
x551[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x464, x464));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    32, 128, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x542, x542));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x551, in_desc, x533, filt_desc, x125,
    conv_desc, algo, ws_data, ws_size,
    x549, out_desc, x548));
};
float* x554 = (float*)myMalloc(1 * sizeof(float));;
x554[0] = 1.0f;
float* x556 = (float*)myMalloc(1 * sizeof(float));;
x556[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 32, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x542, x542));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x554, bias_desc, x164, x556, out_desc, x548));
};
float* x559 = (float*)myMalloc(1 * sizeof(float));;
x559[0] = 0.0f;
float* x561 = (float*)myMalloc(1 * sizeof(float));;
x561[0] = 1.0f;
float* x563 = (float*)myGpuMalloc(x545 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x542, x542));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x561, x_desc, x548, x559, x_desc, x563));
};
if (x566) {
} else {
assert(false && "ERROR not specified");
}
float* x578 = (float*)myGpuMalloc(x577 * sizeof(float));
float* x579 = (float*)myMalloc(1 * sizeof(float));;
x579[0] = 0.0f;
float* x581 = (float*)myMalloc(1 * sizeof(float));;
x581[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x542, x542));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    128, 32, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x572, x572));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x581, in_desc, x563, filt_desc, x200,
    conv_desc, algo, ws_data, ws_size,
    x579, out_desc, x578));
};
float* x584 = (float*)myMalloc(1 * sizeof(float));;
x584[0] = 1.0f;
float* x586 = (float*)myMalloc(1 * sizeof(float));;
x586[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 128, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x572, x572));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x584, bias_desc, x230, x586, out_desc, x578));
};
float* x589 = (float*)myMalloc(1 * sizeof(float));;
x589[0] = 0.0f;
float* x591 = (float*)myMalloc(1 * sizeof(float));;
x591[0] = 1.0f;
float* x593 = (float*)myGpuMalloc(x575 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x572, x572));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x591, x_desc, x578, x589, x_desc, x593));
};
if (x597) {
} else {
assert(false && "ERROR not specified");
}
float* x610 = (float*)myGpuMalloc(x609 * sizeof(float));
float* x611 = (float*)myMalloc(1 * sizeof(float));;
x611[0] = 0.0f;
float* x613 = (float*)myMalloc(1 * sizeof(float));;
x613[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x542, x542));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    128, 32, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x604, x604));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x613, in_desc, x563, filt_desc, x113,
    conv_desc, algo, ws_data, ws_size,
    x611, out_desc, x610));
};
float* x616 = (float*)myMalloc(1 * sizeof(float));;
x616[0] = 1.0f;
float* x618 = (float*)myMalloc(1 * sizeof(float));;
x618[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 128, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x604, x604));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x616, bias_desc, x218, x618, out_desc, x610));
};
float* x621 = (float*)myMalloc(1 * sizeof(float));;
x621[0] = 0.0f;
float* x623 = (float*)myMalloc(1 * sizeof(float));;
x623[0] = 1.0f;
float* x625 = (float*)myGpuMalloc(x607 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x604, x604));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x623, x_desc, x610, x621, x_desc, x625));
};
if (x630) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x641 = (float*)myGpuMalloc(x640 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x593, 128, x575, x625, 128, x607, x641, 1, 64, 256, x572, x572, x637, x573, x572, 1);
};
float* x643 = (float*)myMalloc(1 * sizeof(float));;
x643[0] = 0.0f;
float* x645 = (float*)myMalloc(1 * sizeof(float));;
x645[0] = 1.0f;
float* x655 = (float*)myGpuMalloc(x654 * sizeof(float));

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x572, x572) );

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x649, x649));

hipdnnPoolingDescriptor_t poolingDesc;
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&poolingDesc));
CUDNN_CALL(hipdnnSetPooling2dDescriptor(
    poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
    2, 2, 0,
    0, 2, 2
));
CUDNN_CALL(hipdnnPoolingForward(
    cudnnHandle, 
    poolingDesc, 
    x645, in_desc, x641, x643, out_desc, x655));
};
if (x658) {
} else {
assert(false && "ERROR not specified");
}
float* x670 = (float*)myGpuMalloc(x669 * sizeof(float));
float* x671 = (float*)myMalloc(1 * sizeof(float));;
x671[0] = 0.0f;
float* x673 = (float*)myMalloc(1 * sizeof(float));;
x673[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x649, x649));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    32, 256, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x664, x664));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x673, in_desc, x655, filt_desc, x176,
    conv_desc, algo, ws_data, ws_size,
    x671, out_desc, x670));
};
float* x676 = (float*)myMalloc(1 * sizeof(float));;
x676[0] = 1.0f;
float* x678 = (float*)myMalloc(1 * sizeof(float));;
x678[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 32, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x664, x664));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x676, bias_desc, x140, x678, out_desc, x670));
};
float* x681 = (float*)myMalloc(1 * sizeof(float));;
x681[0] = 0.0f;
float* x683 = (float*)myMalloc(1 * sizeof(float));;
x683[0] = 1.0f;
float* x685 = (float*)myGpuMalloc(x667 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x664, x664));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x683, x_desc, x670, x681, x_desc, x685));
};
if (x688) {
} else {
assert(false && "ERROR not specified");
}
float* x700 = (float*)myGpuMalloc(x699 * sizeof(float));
float* x701 = (float*)myMalloc(1 * sizeof(float));;
x701[0] = 0.0f;
float* x703 = (float*)myMalloc(1 * sizeof(float));;
x703[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x664, x664));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    128, 32, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x694, x694));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x703, in_desc, x685, filt_desc, x116,
    conv_desc, algo, ws_data, ws_size,
    x701, out_desc, x700));
};
float* x706 = (float*)myMalloc(1 * sizeof(float));;
x706[0] = 1.0f;
float* x708 = (float*)myMalloc(1 * sizeof(float));;
x708[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 128, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x694, x694));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x706, bias_desc, x158, x708, out_desc, x700));
};
float* x711 = (float*)myMalloc(1 * sizeof(float));;
x711[0] = 0.0f;
float* x713 = (float*)myMalloc(1 * sizeof(float));;
x713[0] = 1.0f;
float* x715 = (float*)myGpuMalloc(x697 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x694, x694));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x713, x_desc, x700, x711, x_desc, x715));
};
if (x719) {
} else {
assert(false && "ERROR not specified");
}
float* x732 = (float*)myGpuMalloc(x731 * sizeof(float));
float* x733 = (float*)myMalloc(1 * sizeof(float));;
x733[0] = 0.0f;
float* x735 = (float*)myMalloc(1 * sizeof(float));;
x735[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 32, x664, x664));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    128, 32, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x726, x726));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x735, in_desc, x685, filt_desc, x203,
    conv_desc, algo, ws_data, ws_size,
    x733, out_desc, x732));
};
float* x738 = (float*)myMalloc(1 * sizeof(float));;
x738[0] = 1.0f;
float* x740 = (float*)myMalloc(1 * sizeof(float));;
x740[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 128, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x726, x726));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x738, bias_desc, x143, x740, out_desc, x732));
};
float* x743 = (float*)myMalloc(1 * sizeof(float));;
x743[0] = 0.0f;
float* x745 = (float*)myMalloc(1 * sizeof(float));;
x745[0] = 1.0f;
float* x747 = (float*)myGpuMalloc(x729 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 128, x726, x726));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x745, x_desc, x732, x743, x_desc, x747));
};
if (x752) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x763 = (float*)myGpuMalloc(x762 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x715, 128, x697, x747, 128, x729, x763, 1, 64, 256, x694, x694, x759, x695, x694, 1);
};
if (x766) {
} else {
assert(false && "ERROR not specified");
}
float* x778 = (float*)myGpuMalloc(x777 * sizeof(float));
float* x779 = (float*)myMalloc(1 * sizeof(float));;
x779[0] = 0.0f;
float* x781 = (float*)myMalloc(1 * sizeof(float));;
x781[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x694, x694));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    48, 256, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x772, x772));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x781, in_desc, x763, filt_desc, x221,
    conv_desc, algo, ws_data, ws_size,
    x779, out_desc, x778));
};
float* x784 = (float*)myMalloc(1 * sizeof(float));;
x784[0] = 1.0f;
float* x786 = (float*)myMalloc(1 * sizeof(float));;
x786[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 48, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x772, x772));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x784, bias_desc, x251, x786, out_desc, x778));
};
float* x789 = (float*)myMalloc(1 * sizeof(float));;
x789[0] = 0.0f;
float* x791 = (float*)myMalloc(1 * sizeof(float));;
x791[0] = 1.0f;
float* x793 = (float*)myGpuMalloc(x775 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x772, x772));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x791, x_desc, x778, x789, x_desc, x793));
};
if (x796) {
} else {
assert(false && "ERROR not specified");
}
float* x808 = (float*)myGpuMalloc(x807 * sizeof(float));
float* x809 = (float*)myMalloc(1 * sizeof(float));;
x809[0] = 0.0f;
float* x811 = (float*)myMalloc(1 * sizeof(float));;
x811[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x772, x772));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    192, 48, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x802, x802));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x811, in_desc, x793, filt_desc, x239,
    conv_desc, algo, ws_data, ws_size,
    x809, out_desc, x808));
};
float* x814 = (float*)myMalloc(1 * sizeof(float));;
x814[0] = 1.0f;
float* x816 = (float*)myMalloc(1 * sizeof(float));;
x816[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 192, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x802, x802));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x814, bias_desc, x233, x816, out_desc, x808));
};
float* x819 = (float*)myMalloc(1 * sizeof(float));;
x819[0] = 0.0f;
float* x821 = (float*)myMalloc(1 * sizeof(float));;
x821[0] = 1.0f;
float* x823 = (float*)myGpuMalloc(x805 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x802, x802));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x821, x_desc, x808, x819, x_desc, x823));
};
if (x827) {
} else {
assert(false && "ERROR not specified");
}
float* x840 = (float*)myGpuMalloc(x839 * sizeof(float));
float* x841 = (float*)myMalloc(1 * sizeof(float));;
x841[0] = 0.0f;
float* x843 = (float*)myMalloc(1 * sizeof(float));;
x843[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x772, x772));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    192, 48, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x834, x834));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x843, in_desc, x793, filt_desc, x212,
    conv_desc, algo, ws_data, ws_size,
    x841, out_desc, x840));
};
float* x846 = (float*)myMalloc(1 * sizeof(float));;
x846[0] = 1.0f;
float* x848 = (float*)myMalloc(1 * sizeof(float));;
x848[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 192, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x834, x834));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x846, bias_desc, x182, x848, out_desc, x840));
};
float* x851 = (float*)myMalloc(1 * sizeof(float));;
x851[0] = 0.0f;
float* x853 = (float*)myMalloc(1 * sizeof(float));;
x853[0] = 1.0f;
float* x855 = (float*)myGpuMalloc(x837 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x834, x834));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x853, x_desc, x840, x851, x_desc, x855));
};
if (x860) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x871 = (float*)myGpuMalloc(x870 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x823, 192, x805, x855, 192, x837, x871, 1, 64, 384, x802, x802, x867, x803, x802, 1);
};
if (x874) {
} else {
assert(false && "ERROR not specified");
}
float* x886 = (float*)myGpuMalloc(x885 * sizeof(float));
float* x887 = (float*)myMalloc(1 * sizeof(float));;
x887[0] = 0.0f;
float* x889 = (float*)myMalloc(1 * sizeof(float));;
x889[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 384, x802, x802));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    48, 384, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x880, x880));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x889, in_desc, x871, filt_desc, x137,
    conv_desc, algo, ws_data, ws_size,
    x887, out_desc, x886));
};
float* x892 = (float*)myMalloc(1 * sizeof(float));;
x892[0] = 1.0f;
float* x894 = (float*)myMalloc(1 * sizeof(float));;
x894[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 48, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x880, x880));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x892, bias_desc, x101, x894, out_desc, x886));
};
float* x897 = (float*)myMalloc(1 * sizeof(float));;
x897[0] = 0.0f;
float* x899 = (float*)myMalloc(1 * sizeof(float));;
x899[0] = 1.0f;
float* x901 = (float*)myGpuMalloc(x883 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x880, x880));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x899, x_desc, x886, x897, x_desc, x901));
};
if (x904) {
} else {
assert(false && "ERROR not specified");
}
float* x916 = (float*)myGpuMalloc(x915 * sizeof(float));
float* x917 = (float*)myMalloc(1 * sizeof(float));;
x917[0] = 0.0f;
float* x919 = (float*)myMalloc(1 * sizeof(float));;
x919[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x880, x880));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    192, 48, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x910, x910));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x919, in_desc, x901, filt_desc, x161,
    conv_desc, algo, ws_data, ws_size,
    x917, out_desc, x916));
};
float* x922 = (float*)myMalloc(1 * sizeof(float));;
x922[0] = 1.0f;
float* x924 = (float*)myMalloc(1 * sizeof(float));;
x924[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 192, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x910, x910));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x922, bias_desc, x191, x924, out_desc, x916));
};
float* x927 = (float*)myMalloc(1 * sizeof(float));;
x927[0] = 0.0f;
float* x929 = (float*)myMalloc(1 * sizeof(float));;
x929[0] = 1.0f;
float* x931 = (float*)myGpuMalloc(x913 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x910, x910));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x929, x_desc, x916, x927, x_desc, x931));
};
if (x935) {
} else {
assert(false && "ERROR not specified");
}
float* x948 = (float*)myGpuMalloc(x947 * sizeof(float));
float* x949 = (float*)myMalloc(1 * sizeof(float));;
x949[0] = 0.0f;
float* x951 = (float*)myMalloc(1 * sizeof(float));;
x951[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 48, x880, x880));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    192, 48, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x942, x942));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x951, in_desc, x901, filt_desc, x149,
    conv_desc, algo, ws_data, ws_size,
    x949, out_desc, x948));
};
float* x954 = (float*)myMalloc(1 * sizeof(float));;
x954[0] = 1.0f;
float* x956 = (float*)myMalloc(1 * sizeof(float));;
x956[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 192, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x942, x942));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x954, bias_desc, x227, x956, out_desc, x948));
};
float* x959 = (float*)myMalloc(1 * sizeof(float));;
x959[0] = 0.0f;
float* x961 = (float*)myMalloc(1 * sizeof(float));;
x961[0] = 1.0f;
float* x963 = (float*)myGpuMalloc(x945 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 192, x942, x942));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x961, x_desc, x948, x959, x_desc, x963));
};
if (x968) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x979 = (float*)myGpuMalloc(x978 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x931, 192, x913, x963, 192, x945, x979, 1, 64, 384, x910, x910, x975, x911, x910, 1);
};
if (x982) {
} else {
assert(false && "ERROR not specified");
}
float* x994 = (float*)myGpuMalloc(x993 * sizeof(float));
float* x995 = (float*)myMalloc(1 * sizeof(float));;
x995[0] = 0.0f;
float* x997 = (float*)myMalloc(1 * sizeof(float));;
x997[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 384, x910, x910));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    64, 384, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x988, x988));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x997, in_desc, x979, filt_desc, x197,
    conv_desc, algo, ws_data, ws_size,
    x995, out_desc, x994));
};
float* x1000 = (float*)myMalloc(1 * sizeof(float));;
x1000[0] = 1.0f;
float* x1002 = (float*)myMalloc(1 * sizeof(float));;
x1002[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x988, x988));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x1000, bias_desc, x122, x1002, out_desc, x994));
};
float* x1005 = (float*)myMalloc(1 * sizeof(float));;
x1005[0] = 0.0f;
float* x1007 = (float*)myMalloc(1 * sizeof(float));;
x1007[0] = 1.0f;
float* x1009 = (float*)myGpuMalloc(x991 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x988, x988));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x1007, x_desc, x994, x1005, x_desc, x1009));
};
if (x1012) {
} else {
assert(false && "ERROR not specified");
}
float* x1024 = (float*)myGpuMalloc(x1023 * sizeof(float));
float* x1025 = (float*)myMalloc(1 * sizeof(float));;
x1025[0] = 0.0f;
float* x1027 = (float*)myMalloc(1 * sizeof(float));;
x1027[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x988, x988));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    256, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1018, x1018));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x1027, in_desc, x1009, filt_desc, x242,
    conv_desc, algo, ws_data, ws_size,
    x1025, out_desc, x1024));
};
float* x1030 = (float*)myMalloc(1 * sizeof(float));;
x1030[0] = 1.0f;
float* x1032 = (float*)myMalloc(1 * sizeof(float));;
x1032[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 256, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1018, x1018));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x1030, bias_desc, x215, x1032, out_desc, x1024));
};
float* x1035 = (float*)myMalloc(1 * sizeof(float));;
x1035[0] = 0.0f;
float* x1037 = (float*)myMalloc(1 * sizeof(float));;
x1037[0] = 1.0f;
float* x1039 = (float*)myGpuMalloc(x1021 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1018, x1018));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x1037, x_desc, x1024, x1035, x_desc, x1039));
};
if (x1043) {
} else {
assert(false && "ERROR not specified");
}
float* x1056 = (float*)myGpuMalloc(x1055 * sizeof(float));
float* x1057 = (float*)myMalloc(1 * sizeof(float));;
x1057[0] = 0.0f;
float* x1059 = (float*)myMalloc(1 * sizeof(float));;
x1059[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x988, x988));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    256, 64, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1050, x1050));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x1059, in_desc, x1009, filt_desc, x179,
    conv_desc, algo, ws_data, ws_size,
    x1057, out_desc, x1056));
};
float* x1062 = (float*)myMalloc(1 * sizeof(float));;
x1062[0] = 1.0f;
float* x1064 = (float*)myMalloc(1 * sizeof(float));;
x1064[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 256, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1050, x1050));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x1062, bias_desc, x134, x1064, out_desc, x1056));
};
float* x1067 = (float*)myMalloc(1 * sizeof(float));;
x1067[0] = 0.0f;
float* x1069 = (float*)myMalloc(1 * sizeof(float));;
x1069[0] = 1.0f;
float* x1071 = (float*)myGpuMalloc(x1053 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1050, x1050));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x1069, x_desc, x1056, x1067, x_desc, x1071));
};
if (x1076) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x1087 = (float*)myGpuMalloc(x1086 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x1039, 256, x1021, x1071, 256, x1053, x1087, 1, 64, 512, x1018, x1018, x1083, x1019, x1018, 1);
};
float* x1089 = (float*)myMalloc(1 * sizeof(float));;
x1089[0] = 0.0f;
float* x1091 = (float*)myMalloc(1 * sizeof(float));;
x1091[0] = 1.0f;
float* x1101 = (float*)myGpuMalloc(x1100 * sizeof(float));

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 512, x1018, x1018) );

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 512, x1095, x1095));

hipdnnPoolingDescriptor_t poolingDesc;
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&poolingDesc));
CUDNN_CALL(hipdnnSetPooling2dDescriptor(
    poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_NOT_PROPAGATE_NAN,
    2, 2, 0,
    0, 2, 2
));
CUDNN_CALL(hipdnnPoolingForward(
    cudnnHandle, 
    poolingDesc, 
    x1091, in_desc, x1087, x1089, out_desc, x1101));
};
if (x1104) {
} else {
assert(false && "ERROR not specified");
}
float* x1116 = (float*)myGpuMalloc(x1115 * sizeof(float));
float* x1117 = (float*)myMalloc(1 * sizeof(float));;
x1117[0] = 0.0f;
float* x1119 = (float*)myMalloc(1 * sizeof(float));;
x1119[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 512, x1095, x1095));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    64, 512, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x1110, x1110));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x1119, in_desc, x1101, filt_desc, x98,
    conv_desc, algo, ws_data, ws_size,
    x1117, out_desc, x1116));
};
float* x1122 = (float*)myMalloc(1 * sizeof(float));;
x1122[0] = 1.0f;
float* x1124 = (float*)myMalloc(1 * sizeof(float));;
x1124[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x1110, x1110));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x1122, bias_desc, x155, x1124, out_desc, x1116));
};
float* x1127 = (float*)myMalloc(1 * sizeof(float));;
x1127[0] = 0.0f;
float* x1129 = (float*)myMalloc(1 * sizeof(float));;
x1129[0] = 1.0f;
float* x1131 = (float*)myGpuMalloc(x1113 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x1110, x1110));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x1129, x_desc, x1116, x1127, x_desc, x1131));
};
if (x1134) {
} else {
assert(false && "ERROR not specified");
}
float* x1146 = (float*)myGpuMalloc(x1145 * sizeof(float));
float* x1147 = (float*)myMalloc(1 * sizeof(float));;
x1147[0] = 0.0f;
float* x1149 = (float*)myMalloc(1 * sizeof(float));;
x1149[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x1110, x1110));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    256, 64, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1140, x1140));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x1149, in_desc, x1131, filt_desc, x209,
    conv_desc, algo, ws_data, ws_size,
    x1147, out_desc, x1146));
};
float* x1152 = (float*)myMalloc(1 * sizeof(float));;
x1152[0] = 1.0f;
float* x1154 = (float*)myMalloc(1 * sizeof(float));;
x1154[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 256, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1140, x1140));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x1152, bias_desc, x173, x1154, out_desc, x1146));
};
float* x1157 = (float*)myMalloc(1 * sizeof(float));;
x1157[0] = 0.0f;
float* x1159 = (float*)myMalloc(1 * sizeof(float));;
x1159[0] = 1.0f;
float* x1161 = (float*)myGpuMalloc(x1143 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1140, x1140));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x1159, x_desc, x1146, x1157, x_desc, x1161));
};
if (x1165) {
} else {
assert(false && "ERROR not specified");
}
float* x1178 = (float*)myGpuMalloc(x1177 * sizeof(float));
float* x1179 = (float*)myMalloc(1 * sizeof(float));;
x1179[0] = 0.0f;
float* x1181 = (float*)myMalloc(1 * sizeof(float));;
x1181[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 64, x1110, x1110));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    256, 64, 3, 3));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1172, x1172));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    1, 1, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x1181, in_desc, x1131, filt_desc, x185,
    conv_desc, algo, ws_data, ws_size,
    x1179, out_desc, x1178));
};
float* x1184 = (float*)myMalloc(1 * sizeof(float));;
x1184[0] = 1.0f;
float* x1186 = (float*)myMalloc(1 * sizeof(float));;
x1186[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 256, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1172, x1172));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x1184, bias_desc, x146, x1186, out_desc, x1178));
};
float* x1189 = (float*)myMalloc(1 * sizeof(float));;
x1189[0] = 0.0f;
float* x1191 = (float*)myMalloc(1 * sizeof(float));;
x1191[0] = 1.0f;
float* x1193 = (float*)myGpuMalloc(x1175 * sizeof(float));

{
hipdnnTensorDescriptor_t x_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&x_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 256, x1172, x1172));

hipdnnActivationDescriptor_t act_desc;
CUDNN_CALL(hipdnnCreateActivationDescriptor(&act_desc));
CUDNN_CALL(hipdnnSetActivationDescriptor(act_desc,
                                        /*mode=*/ HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/ HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/ 0));
CUDNN_CALL(hipdnnActivationForward(
    cudnnHandle, act_desc,
    x1191, x_desc, x1178, x1189, x_desc, x1193));
};
if (x1198) {
} else {
printf("all dimensions except the concatenation dimension should be the same\n");
assert(false && "");
}
// back prop for concat
float* x1209 = (float*)myGpuMalloc(x1208 * sizeof(float));
{
dim3 grid(28, 2);
concat2D_1D_greg<<<grid, 512>>>(x1161, 256, x1143, x1193, 256, x1175, x1209, 1, 64, 512, x1140, x1140, x1205, x1141, x1140, 1);
};
if (x1212) {
} else {
assert(false && "ERROR not specified");
}
float* x1225 = (float*)myGpuMalloc(x1224 * sizeof(float));
float* x1226 = (float*)myMalloc(1 * sizeof(float));;
x1226[0] = 0.0f;
float* x1228 = (float*)myMalloc(1 * sizeof(float));;
x1228[0] = 1.0f;

{
hipdnnTensorDescriptor_t in_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 512, x1140, x1140));

hipdnnFilterDescriptor_t filt_desc;
CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
    10, 512, 4, 4));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 10, x1219, x1219));

hipdnnConvolutionDescriptor_t conv_desc;
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc,
    0, 0, 1, 1, 1, 1,
    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
// Algorithm.
hipdnnConvolutionFwdAlgo_t algo;
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    cudnnHandle,
    in_desc, filt_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

// Workspace.
size_t ws_size;
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    cudnnHandle, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
void *ws_data = myGpuMalloc(ws_size);
// Execute convolution.
CUDNN_CALL(hipdnnConvolutionForward(
    cudnnHandle,
    x1228, in_desc, x1209, filt_desc, x107,
    conv_desc, algo, ws_data, ws_size,
    x1226, out_desc, x1225));
};
float* x1231 = (float*)myMalloc(1 * sizeof(float));;
x1231[0] = 1.0f;
float* x1233 = (float*)myMalloc(1 * sizeof(float));;
x1233[0] = 1.0f;

{
hipdnnTensorDescriptor_t bias_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    1, 10, 1, 1));

hipdnnTensorDescriptor_t out_desc;
CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    64, 10, x1219, x1219));

CUDNN_CALL(hipdnnAddTensor(
    cudnnHandle, x1231, bias_desc, x248, x1233, out_desc, x1225));
};
int64_t x1236 = (long)mallocAddr;
int64_t x1237 = x1236 - x253;
memset((void*)x253, 0, x1237);
mallocAddr = (void*)x253;
int64_t x1240 = (long)gpuMallocAddr;
int64_t x1241 = x1240 - x254;
hipMemset((void*)x254, 0, x1241);
gpuMallocAddr = (void*)x254;

}
gettimeofday(&end_1, NULL);
timeval_subtract(&diff_1, &end_1, &begin_1);;
int64_t x1248 = ((diff_1.tv_sec * 1000000L) + (diff_1.tv_usec));
int64_t x1249 = x1248 / 1000LL;
int64_t x1251 = x1248 / x1250;
printf("Inferencing completed in %ldms (%ld us/images)\n",x1249,x1251);

}
// Backend cleanup.
CUBLAS_CALL(hipblasDestroy(cublasHandle));
CUDA_CALL(hipFree(gpuMallocBase));
      
CUDNN_CALL(hipdnnDestroy(cudnnHandle));
}
/*****************************************
  End of C Generated Code                  
*******************************************/

